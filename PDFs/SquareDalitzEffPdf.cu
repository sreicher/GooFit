#include "hip/hip_runtime.h"
#include "SquareDalitzEffPdf.hh"

EXEC_TARGET fptype inPS(fptype m12, fptype m13, fptype mD, fptype mKS0, fptype mh1, fptype mh2) {

  if (m12 < pow(mKS0 + mh1, 2)) return 0;
  if (m12 > pow(mD - mh2, 2)) return 0;

  // Calculate energies of 1 and 3 particles in m12 rest frame. 
  fptype e1star = 0.5 * (m12 - mh1*mh1 + mKS0*mKS0) / sqrt(m12);
  fptype e3star = 0.5 * (mD*mD - m12 - mh2*mh2) / sqrt(m12);

  fptype minimum = pow(e1star + e3star, 2) - pow(sqrt(e1star*e1star - mKS0*mKS0) + sqrt(e3star*e3star - mh2*mh2), 2);
  if (m13 < minimum) return 0;
  fptype maximum = pow(e1star + e3star, 2) - pow(sqrt(e1star*e1star - mKS0*mKS0) - sqrt(e3star*e3star - mh2*mh2), 2);
  if (m13 > maximum) return 0;

  return 1;
}

EXEC_TARGET fptype mprime (fptype m12, fptype m13, fptype mD, fptype mKS0, fptype mh1, fptype mh2) {
  // Helper function to calculate m'^2
  fptype m23 = mD*mD + mKS0*mKS0 + mh1*mh1 + mh2*mh2 - m12 - m13; 
  fptype rootPi = -2.*ATAN2(-1.0,0.0); // Pi

  if (m23 < 0) return -99;
  fptype tmp = ((2.0*(SQRT(m23) - (mh1 + mh2))/(mD - mKS0 - (mh1 + mh2))) - 1.0);
  if (isnan(tmp)) tmp = -99;
  return tmp;
}

EXEC_TARGET fptype thetaprime (fptype m12, fptype m13, fptype mD, fptype mKS0, fptype mh1, fptype mh2) {
  // Helper function to calculate theta'
  fptype m23 = mD*mD + mKS0*mKS0 + mh1*mh1 + mh2*mh2 - m12 - m13; 
  if (m23 < 0) return -99;

  fptype num = m23*( m12 - m13) + (mh2*mh2 - mh1*mh1)*(mD*mD - mKS0*mKS0);
  fptype denum = SQRT(((m23 - mh1*mh1 + mh2*mh2)*(m23 - mh1*mh1 + mh2*mh2) - 4*m23*mh2*mh2))*SQRT(((mD*mD - mKS0*mKS0 - m23)*(mD*mD - mKS0*mKS0 -m23) - 4*m23*mKS0*mKS0));
  fptype theta = -99 ;
  if (isnan(denum)) return -99;

  if (denum != 0.){
    theta = num/denum;
  }

  return theta;
}

EXEC_TARGET fptype device_SquareDalitzEff (fptype* evt, fptype* p, unsigned int* indices) {
  // Implementation of m23, mprime, thetaprime and also of efficiency tested
  // eff =  [0]*pow(theta,2) + [1]*m*pow(theta,2) +  exp([2] + [3]*dtime + ([4] + [5]*dtime)*m)

  // Define observables 
  fptype x = evt[indices[2 + indices[0] + 0]]; // m12   
  fptype y = evt[indices[2 + indices[0] + 1]]; // m13   
  //fptype z = evt[indices[2 + indices[0] + 2]]; // dtime   

  // Define constvals
  fptype mD   = p[indices[1]];
  fptype mKS0 = p[indices[2]];
  fptype mh1  = p[indices[3]];
  fptype mh2  = p[indices[4]];

  // Define coefficients
  fptype c0 = p[indices[5]];   
  fptype c1 = p[indices[6]];   
  fptype c2 = p[indices[7]];   
  fptype c3 = p[indices[8]];   
  fptype c4 = p[indices[9]];   
  fptype c5 = p[indices[10]];   
  // Check phase space
  if (inPS == 0) return 0;
  
  // Call helper functions
  fptype thetap = thetaprime(x,y,mD,mKS0,mh1,mh2); 
  if (thetap > 1. || thetap < -1.) return 0; 

  fptype m23 = mD*mD + mKS0*mKS0 + mh1*mh1 + mh2*mh2 - x - y; 
  if (m23 < 0) return 0;

  fptype ret = c0*m23*m23 + c1*m23 + c2*m23*thetap*thetap + c3*thetap*thetap + c4*thetap + c5; 

  //fptype mp     = mprime(x,y,mD,mKS0,mh1,mh2); 
  //if (mp > 1. || mp < -1.) return 0;
  //fptype rootPi = -2.*ATAN2(-1.0,0.0); // Pi
  // Calculate acutual m'^2 and theta'
  //mp     = POW(ACOS( mp )/rootPi,2);
  //thetap = ACOS( thetap )/rootPi;

  //fptype tmp = c2 + c3*mp + c4*POW(mp,2);
  //fptype ret = c0*POW(thetap,2) + c1*mp*POW(thetap,2) + tmp;
  //printf("Efficiency %f and m12 %f and m13 %f and dtime %f\n", ret, x, y, z);
  //printf("Efficiency %f and mp %f and thetap %f and dtime %f\n", ret, mp, thetap, z);
  
  return ret; 
}

MEM_DEVICE device_function_ptr ptr_to_SquareDalitzEff = device_SquareDalitzEff; 

__host__ SquareDalitzEffPdf::SquareDalitzEffPdf (std::string n, vector<Variable*> obses, vector<Variable*> coeffs, vector<Variable*> constvals) 
  : GooPdf(0, n) 
{
  // Register observables - here m12, m13 and dtime
  for (unsigned int i = 0; i < obses.size(); ++i) {
    registerObservable(obses[i]);
  }

  std::vector<unsigned int> pindices;
  // Register constvals
  for (vector<Variable*>::iterator v = constvals.begin(); v != constvals.end(); ++v) {
    pindices.push_back(registerParameter(*v));
  }
  // Register coefficients
  for (vector<Variable*>::iterator c = coeffs.begin(); c != coeffs.end(); ++c) {
    pindices.push_back(registerParameter(*c));
  }

  GET_FUNCTION_ADDR(ptr_to_SquareDalitzEff);
  initialise(pindices);
}

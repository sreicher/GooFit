#include "hip/hip_runtime.h"
#include "ResonancePdf.hh" 

EXEC_TARGET fptype twoBodyCMmom (fptype rMassSq, fptype d1m, fptype d2m) {
  // For A -> B + C, calculate momentum of B and C in rest frame of A. 
  // PDG 38.16.

  fptype kin1 = 1 - POW(d1m+d2m, 2) / rMassSq;
  if (kin1 >= 0) kin1 = SQRT(kin1);
  else kin1 = 1;
  fptype kin2 = 1 - POW(d1m-d2m, 2) / rMassSq;
  if (kin2 >= 0) kin2 = SQRT(kin2);
  else kin2 = 1; 

  return 0.5*SQRT(rMassSq)*kin1*kin2; 
}


EXEC_TARGET fptype dampingFactorSquare (fptype cmmom, int spin, fptype mRadius) {
  fptype square = mRadius*mRadius*cmmom*cmmom;
  fptype dfsq = 1 + square; // This accounts for spin 1
  if (2 == spin) dfsq += 8 + 2*square + square*square; // Coefficients are 9, 3, 1.   

  // Spin 3 and up not accounted for. 
  return dfsq; 
}

EXEC_TARGET fptype spinFactor (unsigned int spin, fptype motherMass, fptype daug1Mass, fptype daug2Mass, fptype daug3Mass, fptype m12, fptype m13, fptype m23, unsigned int cyclic_index) {
  if (0 == spin) return 1; // Should not cause branching since every thread evaluates the same resonance at the same time. 
  /*
  // Copied from BdkDMixDalitzAmp
   
  fptype _mA = (PAIR_12 == cyclic_index ? daug1Mass : (PAIR_13 == cyclic_index ? daug1Mass : daug3Mass)); 
  fptype _mB = (PAIR_12 == cyclic_index ? daug2Mass : (PAIR_13 == cyclic_index ? daug3Mass : daug3Mass)); 
  fptype _mC = (PAIR_12 == cyclic_index ? daug3Mass : (PAIR_13 == cyclic_index ? daug2Mass : daug1Mass)); 
    
  fptype _mAC = (PAIR_12 == cyclic_index ? m13 : (PAIR_13 == cyclic_index ? m12 : m12)); 
  fptype _mBC = (PAIR_12 == cyclic_index ? m23 : (PAIR_13 == cyclic_index ? m23 : m13)); 
  fptype _mAB = (PAIR_12 == cyclic_index ? m12 : (PAIR_13 == cyclic_index ? m13 : m23)); 

  // The above, collapsed into single tests where possible. 
  fptype _mA = (PAIR_13 == cyclic_index ? daug3Mass : daug2Mass);
  fptype _mB = (PAIR_23 == cyclic_index ? daug2Mass : daug1Mass); 
  fptype _mC = (PAIR_12 == cyclic_index ? daug3Mass : (PAIR_13 == cyclic_index ? daug2Mass : daug1Mass)); 

  fptype _mAC = (PAIR_23 == cyclic_index ? m13 : m23);
  fptype _mBC = (PAIR_12 == cyclic_index ? m13 : m12);
  fptype _mAB = (PAIR_12 == cyclic_index ? m12 : (PAIR_13 == cyclic_index ? m13 : m23)); 
  */

  // Copied from EvtDalitzReso, with assumption that pairAng convention matches pipipi0 from EvtD0mixDalitz.
  // Again, all threads should get the same branch. 
  fptype _mA = (PAIR_12 == cyclic_index ? daug1Mass : (PAIR_13 == cyclic_index ? daug1Mass : daug2Mass));
  fptype _mB = (PAIR_12 == cyclic_index ? daug2Mass : (PAIR_13 == cyclic_index ? daug3Mass : daug3Mass));
  fptype _mC = (PAIR_12 == cyclic_index ? daug3Mass : (PAIR_13 == cyclic_index ? daug2Mass : daug1Mass));
  fptype _mAC = (PAIR_12 == cyclic_index ? m13 : (PAIR_13 == cyclic_index ? m12 : m12));
  fptype _mBC = (PAIR_12 == cyclic_index ? m23 : (PAIR_13 == cyclic_index ? m23 : m13));
  fptype _mAB = (PAIR_12 == cyclic_index ? m12 : (PAIR_13 == cyclic_index ? m13 : m23));

 /*
  // Copied from EvtDalitzReso, with assumption that pairAng convention matches pipipi0 from EvtD0mixDalitz.
  // Again, all threads should get the same branch. 
  fptype _mA = (PAIR_12 == cyclic_index ? daug1Mass : (PAIR_13 == cyclic_index ? daug3Mass : daug2Mass));
  fptype _mB = (PAIR_12 == cyclic_index ? daug2Mass : (PAIR_13 == cyclic_index ? daug1Mass : daug3Mass));
  fptype _mC = (PAIR_12 == cyclic_index ? daug3Mass : (PAIR_13 == cyclic_index ? daug2Mass : daug1Mass));
  fptype _mAC = (PAIR_12 == cyclic_index ? m13 : (PAIR_13 == cyclic_index ? m23 : m12)); 
  fptype _mBC = (PAIR_12 == cyclic_index ? m23 : (PAIR_13 == cyclic_index ? m12 : m13)); 
  fptype _mAB = (PAIR_12 == cyclic_index ? m12 : (PAIR_13 == cyclic_index ? m13 : m23)); 
*/

  fptype massFactor = 1.0/_mAB;
  fptype sFactor = -1; 
  sFactor *= ((_mBC - _mAC) + (massFactor*(motherMass*motherMass - _mC*_mC)*(_mA*_mA-_mB*_mB)));
  if (2 == spin) {
    sFactor *= sFactor; 
    fptype extraterm = ((_mAB-(2*motherMass*motherMass)-(2*_mC*_mC))+massFactor*pow((motherMass*motherMass-_mC*_mC),2));
    extraterm *= ((_mAB-(2*_mA*_mA)-(2*_mB*_mB))+massFactor*pow((_mA*_mA-_mB*_mB),2));
    extraterm /= 3;
    sFactor -= extraterm;
  }
  return sFactor; 
}

EXEC_TARGET devcomplex<fptype> rhoF( fptype mCh, fptype m23 )
{

  fptype rhoSq = 1. - POW( mCh, 2 ) / m23;

  if ( rhoSq >= 0. )
    return devcomplex<fptype>(SQRT( rhoSq ),0);

  devcomplex<fptype> I;
  I.real = 0.0;
  I.imag = SQRT(- rhoSq);

  return I;
}

EXEC_TARGET devcomplex<fptype> rhoFourPiF( fptype mCh, fptype m23)
{
  if ( m23 > 1. )
    return rhoF( 4. * mCh, m23 );

  fptype m4 = POW( m23, 2 );
  fptype m6 = POW( m23, 3 );
  fptype m8 = POW( m23, 4 );

  fptype term = 0.;
  term += 0.00370909 / m4;
  term -= 0.111203 / m23;
  term += 1.2274;
  term -= 6.39017 * m23;
  term += 16.8358 * m4;
  term -= 21.8845 * m6;
  term += 11.3153 * m8;

  return rhoF( 4. * mCh, 1. ) * devcomplex<fptype>(term,0);
}

EXEC_TARGET devcomplex <fptype> Get_kMatrix (fptype m12, fptype m13, fptype m23, unsigned int* indices) {

  fptype Spr0                   = hipArray[indices[2]];
  unsigned int term             = indices[3]; 
  unsigned int spin             = indices[4];
  unsigned int cyclic_index     = indices[5]; 

  assert(term >= 1 && term <= 6);

  // Particle Masses (notation: h = \eta)
  fptype pionMass   = 0.13957018; // PDG: (139.57018 \pm 0.00035) MeV
  fptype pionMassSq = pionMass*pionMass;
  fptype kMass      = 0.497614;   // PDG: (493.677 \pm 0.016) MeV
  fptype hMass      = 0.547853;   // PDG: (547.853 \pm 0.024) MeV
  fptype hprimeMass = 0.95778;    // PDG: (957.78 \pm 0.06) MeV
  
  // Invariant mass squared of resonant particles (pi+ pi- = m23)
  fptype rMassSq = (PAIR_12 == cyclic_index ? m12 : (PAIR_13 == cyclic_index ? m13 : m23));     

  // Pole Masses:           pipi    KK    4pi    hh   hhprime
  fptype poleMassesSq[5] = {0.651,1.2036,1.55817,1.21,1.82206};  // GeV
  for (int m=0; m<=4; m++){ poleMassesSq[m] *= poleMassesSq[m]; } // GeV^2

  // Other Fixed Single Variables
  fptype Ssc_0 = -3.92637; // GeV^2
  fptype sA_0 = -0.15;     // GeV^2
  fptype sA = 1.0;

  // Defining symmetric f^scattering Matrix of elements
  fptype Fsc_matrix[25];
  // non-zero terms;
  Fsc_matrix[0] = 0.23399;
  Fsc_matrix[1] = 0.15044;   Fsc_matrix[5] = 0.15044;
  Fsc_matrix[2] = -0.20545;  Fsc_matrix[10] = -0.20545;
  Fsc_matrix[3] = 0.32825;   Fsc_matrix[15] = 0.32825;
  Fsc_matrix[4] = 0.35412;   Fsc_matrix[20] = 0.35412;

  // all other terms are zero
  for (int i=1; i<=4; i++){
      for (int j=1; j<=4; j++){
          Fsc_matrix[5*i+j]=0.0;
      }
  }

  // Defining g^0 Matrix of elements
  fptype g0_matrix[25];

  g0_matrix[0]  = 0.22889;
  g0_matrix[1]  = -0.55377;
  g0_matrix[2]  = 0.0;
  g0_matrix[3]  = -0.39899;
  g0_matrix[4]  = -0.34639;

  g0_matrix[5]  = 0.94128;
  g0_matrix[6]  = 0.55095;
  g0_matrix[7]  = 0.0;
  g0_matrix[8]  = 0.39065;
  g0_matrix[9]  = 0.31503;

  g0_matrix[10] = 0.36856;
  g0_matrix[11] = 0.23888;
  g0_matrix[12] = 0.55639;
  g0_matrix[13] = 0.1834;
  g0_matrix[14] = 0.18681;

  g0_matrix[15] = 0.3365;
  g0_matrix[16] = 0.40907;
  g0_matrix[17] = 0.85679;
  g0_matrix[18] = 0.19906;
  g0_matrix[19] = -0.00984;

  g0_matrix[20] = 0.18171;
  g0_matrix[21] = -0.17558;
  g0_matrix[22] = -0.79658;
  g0_matrix[23] = -0.00355;
  g0_matrix[24] = 0.22358;

  // Computing the (real and symmetric) K-Matrix
  fptype AdlerZero = (1.0-sA_0)/(rMassSq-sA_0);
         AdlerZero *= (rMassSq - 0.5*sA*pionMassSq);

  devcomplex<fptype> iK_matrix[25]; // i*K-matrix
  for (int i=0; i<=24; i++){ iK_matrix[i] = 0.0;}

  fptype sum_term = 0.0, nonRes_term = 0.0; int irow,jcol,alpha; 
 
  for (irow=0; irow<=4; irow++){
      for (jcol=irow; jcol<=4; jcol++){

	  nonRes_term = Fsc_matrix[5*irow+jcol]*(1.0 - Ssc_0)/(rMassSq - Ssc_0); // Background non-resonant contribution

	  sum_term = 0.0;
	  for (alpha=0; alpha<=4; alpha++){		
	      sum_term += g0_matrix[5*alpha+irow]*g0_matrix[5*alpha+jcol] / (poleMassesSq[alpha]-rMassSq); 
	  }

	 iK_matrix[5*irow + jcol].imag = (sum_term + nonRes_term)*AdlerZero;
	 iK_matrix[5*jcol + irow].imag = (sum_term + nonRes_term)*AdlerZero; // K Matrix is symmetric

      } 
  }
 
  // Calculating pseudo propagator (I - iKp)^-1
  // Calculating Phase Spaces (p = \rho)
  devcomplex<fptype> rho[5];
  for (int d=0; d<=4; d++){rho[d] = 0.0;}
  
  rho[0]    = rhoF(2*pionMass, rMassSq);
  rho[1]      = rhoF(2*kMass, rMassSq);
  rho[2] = rhoFourPiF(pionMass, rMassSq);
  rho[3]      = rhoF(2*hMass, rMassSq);
  rho[4] = rhoF(hMass + hprimeMass, rMassSq);

  // Multiplying iK by diagonal -p matrix
  for (irow=0; irow<=4; irow++){
      for (jcol=0; jcol<=4; jcol++){
          iK_matrix[irow*5+jcol] *= rho[jcol];
          iK_matrix[irow*5+jcol] *= -1;
      }
  }

  // Adding Identity matrix to obtain I-iKp
  for (int d=0; d<=4; d++){
      iK_matrix[d*5+d].real = 1.0 + iK_matrix[d*5+d].real;
  }


  //Explicitly Defining Matrix to be Inverted
  devcomplex <fptype> n11 = iK_matrix[0];
  devcomplex <fptype> n12 = iK_matrix[1];
  devcomplex <fptype> n13 = iK_matrix[2];
  devcomplex <fptype> n14 = iK_matrix[3];
  devcomplex <fptype> n15 = iK_matrix[4];

  devcomplex <fptype> n21 = iK_matrix[5];
  devcomplex <fptype> n22 = iK_matrix[6];
  devcomplex <fptype> n23 = iK_matrix[7];
  devcomplex <fptype> n24 = iK_matrix[8];
  devcomplex <fptype> n25 = iK_matrix[9];

  devcomplex <fptype> n31 = iK_matrix[10];
  devcomplex <fptype> n32 = iK_matrix[11];
  devcomplex <fptype> n33 = iK_matrix[12];
  devcomplex <fptype> n34 = iK_matrix[13];
  devcomplex <fptype> n35 = iK_matrix[14];

  devcomplex <fptype> n41 = iK_matrix[15];
  devcomplex <fptype> n42 = iK_matrix[16];
  devcomplex <fptype> n43 = iK_matrix[17];
  devcomplex <fptype> n44 = iK_matrix[18];
  devcomplex <fptype> n45 = iK_matrix[19];

  devcomplex <fptype> n51 = iK_matrix[20];
  devcomplex <fptype> n52 = iK_matrix[21];
  devcomplex <fptype> n53 = iK_matrix[22];
  devcomplex <fptype> n54 = iK_matrix[23];
  devcomplex <fptype> n55 = iK_matrix[25];

// Computing elements of the first row of (I-iKp)^-1
// Formulae for inverted matrix elements obtained from Maple 

  devcomplex <fptype> inv11(0.0,0.0);
  inv11 += (n22*n33*n44*n55-n22*n33*n45*n54-n22*n34*n43*n55+n22*n34*n45*n53+n22*n35*n43*n54-n22*n35*n44*n53-n23*n32*n44*n55+n23*n32*n45*n54+n23*n34*n42*n55-n23*n34*n45*n52-n23*n35*n42*n54+n23*n35*n44*n52+n24*n32*n43*n55-n24*n32*n45*n53-n24*n33*n42*n55+n24*n33*n45*n52+n24*n35*n42*n53-n24*n35*n43*n52-n25*n32*n43*n54+n25*n32*n44*n53+n25*n33*n42*n54-n25*n33*n44*n52-n25*n34*n42*n53+n25*n34*n43*n52)/(n11*n22*n33*n44*n55-n11*n22*n33*n45*n54-n11*n22*n34*n43*n55+n11*n22*n34*n45*n53+n11*n22*n35*n43*n54-n11*n22*n35*n44*n53-n11*n23*n32*n44*n55+n11*n23*n32*n45*n54+n11*n23*n34*n42*n55-n11*n23*n34*n45*n52-n11*n23*n35*n42*n54+n11*n23*n35*n44*n52+n11*n24*n32*n43*n55-n11*n24*n32*n45*n53-n11*n24*n33*n42*n55+n11*n24*n33*n45*n52+n11*n24*n35*n42*n53-n11*n24*n35*n43*n52-n11*n25*n32*n43*n54+n11*n25*n32*n44*n53+n11*n25*n33*n42*n54-n11*n25*n33*n44*n52-n11*n25*n34*n42*n53+n11*n25*n34*n43*n52-n12*n21*n33*n44*n55+n12*n21*n33*n45*n54+n12*n21*n34*n43*n55-n12*n21*n34*n45*n53-n12*n21*n35*n43*n54+n12*n21*n35*n44*n53+n12*n23*n31*n44*n55-n12*n23*n31*n45*n54-n12*n23*n34*n41*n55+n12*n23*n34*n45*n51+n12*n23*n35*n41*n54-n12*n23*n35*n44*n51-n12*n24*n31*n43*n55+n12*n24*n31*n45*n53+n12*n24*n33*n41*n55-n12*n24*n33*n45*n51-n12*n24*n35*n41*n53+n12*n24*n35*n43*n51+n12*n25*n31*n43*n54-n12*n25*n31*n44*n53-n12*n25*n33*n41*n54+n12*n25*n33*n44*n51+n12*n25*n34*n41*n53-n12*n25*n34*n43*n51+n13*n21*n32*n44*n55-n13*n21*n32*n45*n54-n13*n21*n34*n42*n55+n13*n21*n34*n45*n52+n13*n21*n35*n42*n54-n13*n21*n35*n44*n52-n13*n22*n31*n44*n55+n13*n22*n31*n45*n54+n13*n22*n34*n41*n55-n13*n22*n34*n45*n51-n13*n22*n35*n41*n54+n13*n22*n35*n44*n51+n13*n24*n31*n42*n55-n13*n24*n31*n45*n52-n13*n24*n32*n41*n55+n13*n24*n32*n45*n51+n13*n24*n35*n41*n52-n13*n24*n35*n42*n51-n13*n25*n31*n42*n54+n13*n25*n31*n44*n52+n13*n25*n32*n41*n54-n13*n25*n32*n44*n51-n13*n25*n34*n41*n52+n13*n25*n34*n42*n51-n14*n21*n32*n43*n55+n14*n21*n32*n45*n53+n14*n21*n33*n42*n55-n14*n21*n33*n45*n52-n14*n21*n35*n42*n53+n14*n21*n35*n43*n52+n14*n22*n31*n43*n55-n14*n22*n31*n45*n53-n14*n22*n33*n41*n55+n14*n22*n33*n45*n51+n14*n22*n35*n41*n53-n14*n22*n35*n43*n51-n14*n23*n31*n42*n55+n14*n23*n31*n45*n52+n14*n23*n32*n41*n55-n14*n23*n32*n45*n51-n14*n23*n35*n41*n52+n14*n23*n35*n42*n51+n14*n25*n31*n42*n53-n14*n25*n31*n43*n52-n14*n25*n32*n41*n53+n14*n25*n32*n43*n51+n14*n25*n33*n41*n52-n14*n25*n33*n42*n51+n15*n21*n32*n43*n54-n15*n21*n32*n44*n53-n15*n21*n33*n42*n54+n15*n21*n33*n44*n52+n15*n21*n34*n42*n53-n15*n21*n34*n43*n52-n15*n22*n31*n43*n54+n15*n22*n31*n44*n53+n15*n22*n33*n41*n54-n15*n22*n33*n44*n51-n15*n22*n34*n41*n53+n15*n22*n34*n43*n51+n15*n23*n31*n42*n54-n15*n23*n31*n44*n52-n15*n23*n32*n41*n54+n15*n23*n32*n44*n51+n15*n23*n34*n41*n52-n15*n23*n34*n42*n51-n15*n24*n31*n42*n53+n15*n24*n31*n43*n52+n15*n24*n32*n41*n53-n15*n24*n32*n43*n51-n15*n24*n33*n41*n52+n15*n24*n33*n42*n51); 

  devcomplex <fptype> inv12(0.0,0.0);
  inv12 += -1*(n12*n33*n44*n55-n12*n33*n45*n54-n12*n34*n43*n55+n12*n34*n45*n53+n12*n35*n43*n54-n12*n35*n44*n53-n13*n32*n44*n55+n13*n32*n45*n54+n13*n34*n42*n55-n13*n34*n45*n52-n13*n35*n42*n54+n13*n35*n44*n52+n14*n32*n43*n55-n14*n32*n45*n53-n14*n33*n42*n55+n14*n33*n45*n52+n14*n35*n42*n53-n14*n35*n43*n52-n15*n32*n43*n54+n15*n32*n44*n53+n15*n33*n42*n54-n15*n33*n44*n52-n15*n34*n42*n53+n15*n34*n43*n52)/(n11*n22*n33*n44*n55-n11*n22*n33*n45*n54-n11*n22*n34*n43*n55+n11*n22*n34*n45*n53+n11*n22*n35*n43*n54-n11*n22*n35*n44*n53-n11*n23*n32*n44*n55+n11*n23*n32*n45*n54+n11*n23*n34*n42*n55-n11*n23*n34*n45*n52-n11*n23*n35*n42*n54+n11*n23*n35*n44*n52+n11*n24*n32*n43*n55-n11*n24*n32*n45*n53-n11*n24*n33*n42*n55+n11*n24*n33*n45*n52+n11*n24*n35*n42*n53-n11*n24*n35*n43*n52-n11*n25*n32*n43*n54+n11*n25*n32*n44*n53+n11*n25*n33*n42*n54-n11*n25*n33*n44*n52-n11*n25*n34*n42*n53+n11*n25*n34*n43*n52-n12*n21*n33*n44*n55+n12*n21*n33*n45*n54+n12*n21*n34*n43*n55-n12*n21*n34*n45*n53-n12*n21*n35*n43*n54+n12*n21*n35*n44*n53+n12*n23*n31*n44*n55-n12*n23*n31*n45*n54-n12*n23*n34*n41*n55+n12*n23*n34*n45*n51+n12*n23*n35*n41*n54-n12*n23*n35*n44*n51-n12*n24*n31*n43*n55+n12*n24*n31*n45*n53+n12*n24*n33*n41*n55-n12*n24*n33*n45*n51-n12*n24*n35*n41*n53+n12*n24*n35*n43*n51+n12*n25*n31*n43*n54-n12*n25*n31*n44*n53-n12*n25*n33*n41*n54+n12*n25*n33*n44*n51+n12*n25*n34*n41*n53-n12*n25*n34*n43*n51+n13*n21*n32*n44*n55-n13*n21*n32*n45*n54-n13*n21*n34*n42*n55+n13*n21*n34*n45*n52+n13*n21*n35*n42*n54-n13*n21*n35*n44*n52-n13*n22*n31*n44*n55+n13*n22*n31*n45*n54+n13*n22*n34*n41*n55-n13*n22*n34*n45*n51-n13*n22*n35*n41*n54+n13*n22*n35*n44*n51+n13*n24*n31*n42*n55-n13*n24*n31*n45*n52-n13*n24*n32*n41*n55+n13*n24*n32*n45*n51+n13*n24*n35*n41*n52-n13*n24*n35*n42*n51-n13*n25*n31*n42*n54+n13*n25*n31*n44*n52+n13*n25*n32*n41*n54-n13*n25*n32*n44*n51-n13*n25*n34*n41*n52+n13*n25*n34*n42*n51-n14*n21*n32*n43*n55+n14*n21*n32*n45*n53+n14*n21*n33*n42*n55-n14*n21*n33*n45*n52-n14*n21*n35*n42*n53+n14*n21*n35*n43*n52+n14*n22*n31*n43*n55-n14*n22*n31*n45*n53-n14*n22*n33*n41*n55+n14*n22*n33*n45*n51+n14*n22*n35*n41*n53-n14*n22*n35*n43*n51-n14*n23*n31*n42*n55+n14*n23*n31*n45*n52+n14*n23*n32*n41*n55-n14*n23*n32*n45*n51-n14*n23*n35*n41*n52+n14*n23*n35*n42*n51+n14*n25*n31*n42*n53-n14*n25*n31*n43*n52-n14*n25*n32*n41*n53+n14*n25*n32*n43*n51+n14*n25*n33*n41*n52-n14*n25*n33*n42*n51+n15*n21*n32*n43*n54-n15*n21*n32*n44*n53-n15*n21*n33*n42*n54+n15*n21*n33*n44*n52+n15*n21*n34*n42*n53-n15*n21*n34*n43*n52-n15*n22*n31*n43*n54+n15*n22*n31*n44*n53+n15*n22*n33*n41*n54-n15*n22*n33*n44*n51-n15*n22*n34*n41*n53+n15*n22*n34*n43*n51+n15*n23*n31*n42*n54-n15*n23*n31*n44*n52-n15*n23*n32*n41*n54+n15*n23*n32*n44*n51+n15*n23*n34*n41*n52-n15*n23*n34*n42*n51-n15*n24*n31*n42*n53+n15*n24*n31*n43*n52+n15*n24*n32*n41*n53-n15*n24*n32*n43*n51-n15*n24*n33*n41*n52+n15*n24*n33*n42*n51); 

  devcomplex <fptype> inv13(0.0,0.0);
  inv13 += (n12*n23*n44*n55-n12*n23*n45*n54-n12*n24*n43*n55+n12*n24*n45*n53+n12*n25*n43*n54-n12*n25*n44*n53-n13*n22*n44*n55+n13*n22*n45*n54+n13*n24*n42*n55-n13*n24*n45*n52-n13*n25*n42*n54+n13*n25*n44*n52+n14*n22*n43*n55-n14*n22*n45*n53-n14*n23*n42*n55+n14*n23*n45*n52+n14*n25*n42*n53-n14*n25*n43*n52-n15*n22*n43*n54+n15*n22*n44*n53+n15*n23*n42*n54-n15*n23*n44*n52-n15*n24*n42*n53+n15*n24*n43*n52)/(n11*n22*n33*n44*n55-n11*n22*n33*n45*n54-n11*n22*n34*n43*n55+n11*n22*n34*n45*n53+n11*n22*n35*n43*n54-n11*n22*n35*n44*n53-n11*n23*n32*n44*n55+n11*n23*n32*n45*n54+n11*n23*n34*n42*n55-n11*n23*n34*n45*n52-n11*n23*n35*n42*n54+n11*n23*n35*n44*n52+n11*n24*n32*n43*n55-n11*n24*n32*n45*n53-n11*n24*n33*n42*n55+n11*n24*n33*n45*n52+n11*n24*n35*n42*n53-n11*n24*n35*n43*n52-n11*n25*n32*n43*n54+n11*n25*n32*n44*n53+n11*n25*n33*n42*n54-n11*n25*n33*n44*n52-n11*n25*n34*n42*n53+n11*n25*n34*n43*n52-n12*n21*n33*n44*n55+n12*n21*n33*n45*n54+n12*n21*n34*n43*n55-n12*n21*n34*n45*n53-n12*n21*n35*n43*n54+n12*n21*n35*n44*n53+n12*n23*n31*n44*n55-n12*n23*n31*n45*n54-n12*n23*n34*n41*n55+n12*n23*n34*n45*n51+n12*n23*n35*n41*n54-n12*n23*n35*n44*n51-n12*n24*n31*n43*n55+n12*n24*n31*n45*n53+n12*n24*n33*n41*n55-n12*n24*n33*n45*n51-n12*n24*n35*n41*n53+n12*n24*n35*n43*n51+n12*n25*n31*n43*n54-n12*n25*n31*n44*n53-n12*n25*n33*n41*n54+n12*n25*n33*n44*n51+n12*n25*n34*n41*n53-n12*n25*n34*n43*n51+n13*n21*n32*n44*n55-n13*n21*n32*n45*n54-n13*n21*n34*n42*n55+n13*n21*n34*n45*n52+n13*n21*n35*n42*n54-n13*n21*n35*n44*n52-n13*n22*n31*n44*n55+n13*n22*n31*n45*n54+n13*n22*n34*n41*n55-n13*n22*n34*n45*n51-n13*n22*n35*n41*n54+n13*n22*n35*n44*n51+n13*n24*n31*n42*n55-n13*n24*n31*n45*n52-n13*n24*n32*n41*n55+n13*n24*n32*n45*n51+n13*n24*n35*n41*n52-n13*n24*n35*n42*n51-n13*n25*n31*n42*n54+n13*n25*n31*n44*n52+n13*n25*n32*n41*n54-n13*n25*n32*n44*n51-n13*n25*n34*n41*n52+n13*n25*n34*n42*n51-n14*n21*n32*n43*n55+n14*n21*n32*n45*n53+n14*n21*n33*n42*n55-n14*n21*n33*n45*n52-n14*n21*n35*n42*n53+n14*n21*n35*n43*n52+n14*n22*n31*n43*n55-n14*n22*n31*n45*n53-n14*n22*n33*n41*n55+n14*n22*n33*n45*n51+n14*n22*n35*n41*n53-n14*n22*n35*n43*n51-n14*n23*n31*n42*n55+n14*n23*n31*n45*n52+n14*n23*n32*n41*n55-n14*n23*n32*n45*n51-n14*n23*n35*n41*n52+n14*n23*n35*n42*n51+n14*n25*n31*n42*n53-n14*n25*n31*n43*n52-n14*n25*n32*n41*n53+n14*n25*n32*n43*n51+n14*n25*n33*n41*n52-n14*n25*n33*n42*n51+n15*n21*n32*n43*n54-n15*n21*n32*n44*n53-n15*n21*n33*n42*n54+n15*n21*n33*n44*n52+n15*n21*n34*n42*n53-n15*n21*n34*n43*n52-n15*n22*n31*n43*n54+n15*n22*n31*n44*n53+n15*n22*n33*n41*n54-n15*n22*n33*n44*n51-n15*n22*n34*n41*n53+n15*n22*n34*n43*n51+n15*n23*n31*n42*n54-n15*n23*n31*n44*n52-n15*n23*n32*n41*n54+n15*n23*n32*n44*n51+n15*n23*n34*n41*n52-n15*n23*n34*n42*n51-n15*n24*n31*n42*n53+n15*n24*n31*n43*n52+n15*n24*n32*n41*n53-n15*n24*n32*n43*n51-n15*n24*n33*n41*n52+n15*n24*n33*n42*n51);

  devcomplex <fptype> inv14(0.0,0.0);
  inv14 += -1*(n12*n23*n34*n55-n12*n23*n35*n54-n12*n24*n33*n55+n12*n24*n35*n53+n12*n25*n33*n54-n12*n25*n34*n53-n13*n22*n34*n55+n13*n22*n35*n54+n13*n24*n32*n55-n13*n24*n35*n52-n13*n25*n32*n54+n13*n25*n34*n52+n14*n22*n33*n55-n14*n22*n35*n53-n14*n23*n32*n55+n14*n23*n35*n52+n14*n25*n32*n53-n14*n25*n33*n52-n15*n22*n33*n54+n15*n22*n34*n53+n15*n23*n32*n54-n15*n23*n34*n52-n15*n24*n32*n53+n15*n24*n33*n52)/(n11*n22*n33*n44*n55-n11*n22*n33*n45*n54-n11*n22*n34*n43*n55+n11*n22*n34*n45*n53+n11*n22*n35*n43*n54-n11*n22*n35*n44*n53-n11*n23*n32*n44*n55+n11*n23*n32*n45*n54+n11*n23*n34*n42*n55-n11*n23*n34*n45*n52-n11*n23*n35*n42*n54+n11*n23*n35*n44*n52+n11*n24*n32*n43*n55-n11*n24*n32*n45*n53-n11*n24*n33*n42*n55+n11*n24*n33*n45*n52+n11*n24*n35*n42*n53-n11*n24*n35*n43*n52-n11*n25*n32*n43*n54+n11*n25*n32*n44*n53+n11*n25*n33*n42*n54-n11*n25*n33*n44*n52-n11*n25*n34*n42*n53+n11*n25*n34*n43*n52-n12*n21*n33*n44*n55+n12*n21*n33*n45*n54+n12*n21*n34*n43*n55-n12*n21*n34*n45*n53-n12*n21*n35*n43*n54+n12*n21*n35*n44*n53+n12*n23*n31*n44*n55-n12*n23*n31*n45*n54-n12*n23*n34*n41*n55+n12*n23*n34*n45*n51+n12*n23*n35*n41*n54-n12*n23*n35*n44*n51-n12*n24*n31*n43*n55+n12*n24*n31*n45*n53+n12*n24*n33*n41*n55-n12*n24*n33*n45*n51-n12*n24*n35*n41*n53+n12*n24*n35*n43*n51+n12*n25*n31*n43*n54-n12*n25*n31*n44*n53-n12*n25*n33*n41*n54+n12*n25*n33*n44*n51+n12*n25*n34*n41*n53-n12*n25*n34*n43*n51+n13*n21*n32*n44*n55-n13*n21*n32*n45*n54-n13*n21*n34*n42*n55+n13*n21*n34*n45*n52+n13*n21*n35*n42*n54-n13*n21*n35*n44*n52-n13*n22*n31*n44*n55+n13*n22*n31*n45*n54+n13*n22*n34*n41*n55-n13*n22*n34*n45*n51-n13*n22*n35*n41*n54+n13*n22*n35*n44*n51+n13*n24*n31*n42*n55-n13*n24*n31*n45*n52-n13*n24*n32*n41*n55+n13*n24*n32*n45*n51+n13*n24*n35*n41*n52-n13*n24*n35*n42*n51-n13*n25*n31*n42*n54+n13*n25*n31*n44*n52+n13*n25*n32*n41*n54-n13*n25*n32*n44*n51-n13*n25*n34*n41*n52+n13*n25*n34*n42*n51-n14*n21*n32*n43*n55+n14*n21*n32*n45*n53+n14*n21*n33*n42*n55-n14*n21*n33*n45*n52-n14*n21*n35*n42*n53+n14*n21*n35*n43*n52+n14*n22*n31*n43*n55-n14*n22*n31*n45*n53-n14*n22*n33*n41*n55+n14*n22*n33*n45*n51+n14*n22*n35*n41*n53-n14*n22*n35*n43*n51-n14*n23*n31*n42*n55+n14*n23*n31*n45*n52+n14*n23*n32*n41*n55-n14*n23*n32*n45*n51-n14*n23*n35*n41*n52+n14*n23*n35*n42*n51+n14*n25*n31*n42*n53-n14*n25*n31*n43*n52-n14*n25*n32*n41*n53+n14*n25*n32*n43*n51+n14*n25*n33*n41*n52-n14*n25*n33*n42*n51+n15*n21*n32*n43*n54-n15*n21*n32*n44*n53-n15*n21*n33*n42*n54+n15*n21*n33*n44*n52+n15*n21*n34*n42*n53-n15*n21*n34*n43*n52-n15*n22*n31*n43*n54+n15*n22*n31*n44*n53+n15*n22*n33*n41*n54-n15*n22*n33*n44*n51-n15*n22*n34*n41*n53+n15*n22*n34*n43*n51+n15*n23*n31*n42*n54-n15*n23*n31*n44*n52-n15*n23*n32*n41*n54+n15*n23*n32*n44*n51+n15*n23*n34*n41*n52-n15*n23*n34*n42*n51-n15*n24*n31*n42*n53+n15*n24*n31*n43*n52+n15*n24*n32*n41*n53-n15*n24*n32*n43*n51-n15*n24*n33*n41*n52+n15*n24*n33*n42*n51);

  devcomplex <fptype> inv15(0.0,0.0);
  inv15 += (n12*n23*n34*n45-n12*n23*n35*n44-n12*n24*n33*n45+n12*n24*n35*n43+n12*n25*n33*n44-n12*n25*n34*n43-n13*n22*n34*n45+n13*n22*n35*n44+n13*n24*n32*n45-n13*n24*n35*n42-n13*n25*n32*n44+n13*n25*n34*n42+n14*n22*n33*n45-n14*n22*n35*n43-n14*n23*n32*n45+n14*n23*n35*n42+n14*n25*n32*n43-n14*n25*n33*n42-n15*n22*n33*n44+n15*n22*n34*n43+n15*n23*n32*n44-n15*n23*n34*n42-n15*n24*n32*n43+n15*n24*n33*n42)/(n11*n22*n33*n44*n55-n11*n22*n33*n45*n54-n11*n22*n34*n43*n55+n11*n22*n34*n45*n53+n11*n22*n35*n43*n54-n11*n22*n35*n44*n53-n11*n23*n32*n44*n55+n11*n23*n32*n45*n54+n11*n23*n34*n42*n55-n11*n23*n34*n45*n52-n11*n23*n35*n42*n54+n11*n23*n35*n44*n52+n11*n24*n32*n43*n55-n11*n24*n32*n45*n53-n11*n24*n33*n42*n55+n11*n24*n33*n45*n52+n11*n24*n35*n42*n53-n11*n24*n35*n43*n52-n11*n25*n32*n43*n54+n11*n25*n32*n44*n53+n11*n25*n33*n42*n54-n11*n25*n33*n44*n52-n11*n25*n34*n42*n53+n11*n25*n34*n43*n52-n12*n21*n33*n44*n55+n12*n21*n33*n45*n54+n12*n21*n34*n43*n55-n12*n21*n34*n45*n53-n12*n21*n35*n43*n54+n12*n21*n35*n44*n53+n12*n23*n31*n44*n55-n12*n23*n31*n45*n54-n12*n23*n34*n41*n55+n12*n23*n34*n45*n51+n12*n23*n35*n41*n54-n12*n23*n35*n44*n51-n12*n24*n31*n43*n55+n12*n24*n31*n45*n53+n12*n24*n33*n41*n55-n12*n24*n33*n45*n51-n12*n24*n35*n41*n53+n12*n24*n35*n43*n51+n12*n25*n31*n43*n54-n12*n25*n31*n44*n53-n12*n25*n33*n41*n54+n12*n25*n33*n44*n51+n12*n25*n34*n41*n53-n12*n25*n34*n43*n51+n13*n21*n32*n44*n55-n13*n21*n32*n45*n54-n13*n21*n34*n42*n55+n13*n21*n34*n45*n52+n13*n21*n35*n42*n54-n13*n21*n35*n44*n52-n13*n22*n31*n44*n55+n13*n22*n31*n45*n54+n13*n22*n34*n41*n55-n13*n22*n34*n45*n51-n13*n22*n35*n41*n54+n13*n22*n35*n44*n51+n13*n24*n31*n42*n55-n13*n24*n31*n45*n52-n13*n24*n32*n41*n55+n13*n24*n32*n45*n51+n13*n24*n35*n41*n52-n13*n24*n35*n42*n51-n13*n25*n31*n42*n54+n13*n25*n31*n44*n52+n13*n25*n32*n41*n54-n13*n25*n32*n44*n51-n13*n25*n34*n41*n52+n13*n25*n34*n42*n51-n14*n21*n32*n43*n55+n14*n21*n32*n45*n53+n14*n21*n33*n42*n55-n14*n21*n33*n45*n52-n14*n21*n35*n42*n53+n14*n21*n35*n43*n52+n14*n22*n31*n43*n55-n14*n22*n31*n45*n53-n14*n22*n33*n41*n55+n14*n22*n33*n45*n51+n14*n22*n35*n41*n53-n14*n22*n35*n43*n51-n14*n23*n31*n42*n55+n14*n23*n31*n45*n52+n14*n23*n32*n41*n55-n14*n23*n32*n45*n51-n14*n23*n35*n41*n52+n14*n23*n35*n42*n51+n14*n25*n31*n42*n53-n14*n25*n31*n43*n52-n14*n25*n32*n41*n53+n14*n25*n32*n43*n51+n14*n25*n33*n41*n52-n14*n25*n33*n42*n51+n15*n21*n32*n43*n54-n15*n21*n32*n44*n53-n15*n21*n33*n42*n54+n15*n21*n33*n44*n52+n15*n21*n34*n42*n53-n15*n21*n34*n43*n52-n15*n22*n31*n43*n54+n15*n22*n31*n44*n53+n15*n22*n33*n41*n54-n15*n22*n33*n44*n51-n15*n22*n34*n41*n53+n15*n22*n34*n43*n51+n15*n23*n31*n42*n54-n15*n23*n31*n44*n52-n15*n23*n32*n41*n54+n15*n23*n32*n44*n51+n15*n23*n34*n41*n52-n15*n23*n34*n42*n51-n15*n24*n31*n42*n53+n15*n24*n31*n43*n52+n15*n24*n32*n41*n53-n15*n24*n32*n43*n51-n15*n24*n33*n41*n52+n15*n24*n33*n42*n51); 

  // Computing the F0 element
  fptype temp = (1.0-Spr0)/(rMassSq-Spr0);
  devcomplex <fptype> F0(0.0,0.0);
  devcomplex <fptype> _fr12prod(1.87981, -0.628378);
  devcomplex <fptype> _fr13prod(4.3242, 2.75019);
  devcomplex <fptype> _fr14prod(3.22336, 0.271048);
  devcomplex <fptype> _fr15prod(0., 0.);

  if (term <= 5){ // beta factors
    for (int j = 0; j < 5; j++) {
      fptype tmp = 1./(poleMassesSq[term-1]-rMassSq);
      if (j == 0) F0 += tmp*inv11*g0_matrix[5*(term-1)+j]; 
      else if (j == 1) F0 += tmp*inv12*g0_matrix[5*(term-1)+j]; 
      else if (j == 2) F0 += tmp*inv13*g0_matrix[5*(term-1)+j]; 
      else if (j == 3) F0 += tmp*inv14*g0_matrix[5*(term-1)+j]; 
      else if (j == 4) F0 += tmp*inv15*g0_matrix[5*(term-1)+j]; 
    }
  }
  else { // fprod factors wrt to fprod11
    F0 += inv11;
    F0 += inv12*_fr12prod;
    F0 += inv13*_fr13prod;
    F0 += inv14*_fr14prod;
    F0 += inv15*_fr15prod;
    F0 *= temp;
  }

 return F0;

}

EXEC_TARGET devcomplex<fptype> plainBW (fptype m12, fptype m13, fptype m23, unsigned int* indices) {
  fptype motherMass             = functorConstants[indices[1]+0];
  fptype daug1Mass              = functorConstants[indices[1]+1];
  fptype daug2Mass              = functorConstants[indices[1]+2];
  fptype daug3Mass              = functorConstants[indices[1]+3];
  fptype meson_radius           = functorConstants[indices[1]+4];

  fptype resmass                = hipArray[indices[2]];
  fptype reswidth               = hipArray[indices[3]];
  unsigned int spin             = indices[4];
  unsigned int cyclic_index     = indices[5]; 

  fptype rMassSq = (PAIR_12 == cyclic_index ? m12 : (PAIR_13 == cyclic_index ? m13 : m23));
  fptype frFactor = 1;

  resmass *= resmass; 
  // Calculate momentum of the two daughters in the resonance rest frame; note symmetry under interchange (dm1 <-> dm2). 
  fptype measureDaughterMoms = twoBodyCMmom(rMassSq, 
					    (PAIR_23 == cyclic_index ? daug2Mass : daug1Mass), 
					    (PAIR_12 == cyclic_index ? daug2Mass : daug3Mass));
  fptype nominalDaughterMoms = twoBodyCMmom(resmass, 
					    (PAIR_23 == cyclic_index ? daug2Mass : daug1Mass), 
					    (PAIR_12 == cyclic_index ? daug2Mass : daug3Mass));

  if (0 != spin) {
    frFactor =  dampingFactorSquare(nominalDaughterMoms, spin, meson_radius);
    frFactor /= dampingFactorSquare(measureDaughterMoms, spin, meson_radius); 
  }  
 
  // RBW evaluation
  fptype A = (resmass - rMassSq); 
  fptype B = resmass*reswidth * POW(measureDaughterMoms / nominalDaughterMoms, 2.0*spin + 1) * frFactor / SQRT(rMassSq);
  //fptype C = (SQRT(resmass)*reswidth) / (A*A + B*B); 
  fptype C = 1.0 / (A*A + B*B); 
  devcomplex<fptype> ret(A*C, B*C); // Dropping F_D=1

  ret *= SQRT(frFactor); 
  fptype spinF = spinFactor(spin, motherMass, daug1Mass, daug2Mass, daug3Mass, m12, m13, m23, cyclic_index); 
  ret *= spinF; 

  return ret; 
}

EXEC_TARGET devcomplex<fptype> gaussian (fptype m12, fptype m13, fptype m23, unsigned int* indices) {
  // indices[1] is unused constant index, for consistency with other function types. 
  fptype resmass                = hipArray[indices[2]];
  fptype reswidth               = hipArray[indices[3]];
  unsigned int cyclic_index     = indices[4]; 

  // Notice sqrt - this function uses mass, not mass-squared like the other resonance types. 
  fptype massToUse = SQRT(PAIR_12 == cyclic_index ? m12 : (PAIR_13 == cyclic_index ? m13 : m23));
  massToUse -= resmass;
  massToUse /= reswidth;
  massToUse *= massToUse;
  fptype ret = EXP(-0.5*massToUse); 

  // Ignore factor 1/sqrt(2pi). 
  ret /= reswidth;

  return devcomplex<fptype>(ret, 0); 
}

EXEC_TARGET fptype hFun (fptype s, fptype daug2Mass, fptype daug3Mass) {
  // Last helper function
  const fptype _pi = 3.14159265359;
  fptype sm   = daug2Mass + daug3Mass;
  fptype SQRTs = sqrt(s);
  fptype k_s = twoBodyCMmom(s, daug2Mass, daug3Mass);

  fptype val = ((2/_pi) * (k_s/SQRTs) * log( (SQRTs + 2*k_s)/(sm)));

  return val;
}

EXEC_TARGET fptype dh_dsFun (fptype s, fptype daug2Mass, fptype daug3Mass) {
  // Yet another helper function
  const fptype _pi = 3.14159265359;
  fptype k_s = twoBodyCMmom(s, daug2Mass, daug3Mass);
  
  fptype val = (hFun(s, daug2Mass, daug3Mass) * (1.0/(8.0*pow(k_s, 2)) - 1.0/(2.0 * s)) + 1.0/(2.0* _pi*s));
  return val;
}


EXEC_TARGET fptype dFun (fptype s, fptype daug2Mass, fptype daug3Mass) {
  // Helper function used in Gronau-Sakurai
  const fptype _pi = 3.14159265359;
  fptype sm   = daug2Mass + daug3Mass;
  fptype sm24 = sm*sm/4.0;
  fptype m    = sqrt(s);
  fptype k_m2 = twoBodyCMmom(s, daug2Mass, daug3Mass);
 
  fptype val = 3.0/_pi * sm24/pow(k_m2, 2) * log((m + 2*k_m2)/sm) + m/(2*_pi*k_m2) - sm24*m/(_pi * pow(k_m2, 3));
  return val;
}

EXEC_TARGET fptype fsFun (fptype s, fptype m2, fptype gam, fptype daug2Mass, fptype daug3Mass) {
  // Another G-S helper function
   
  fptype k_s   = twoBodyCMmom(s,  daug2Mass, daug3Mass);
  fptype k_Am2 = twoBodyCMmom(m2, daug2Mass, daug3Mass);
   
  fptype f     = gam * m2 / POW(k_Am2, 3);
  f           *= (POW(k_s, 2) * (hFun(s, daug2Mass, daug3Mass) - hFun(m2, daug2Mass, daug3Mass)) + (m2 - s) * pow(k_Am2, 2) * dh_dsFun(m2, daug2Mass, daug3Mass));
 
  return f;
}

EXEC_TARGET devcomplex<fptype> gouSak (fptype m12, fptype m13, fptype m23, unsigned int* indices) {
  fptype motherMass             = functorConstants[indices[1]+0];
  fptype daug1Mass              = functorConstants[indices[1]+1];
  fptype daug2Mass              = functorConstants[indices[1]+2];
  fptype daug3Mass              = functorConstants[indices[1]+3];
  fptype meson_radius           = functorConstants[indices[1]+4];

  fptype resmass                = hipArray[indices[2]];
  fptype reswidth               = hipArray[indices[3]];
  unsigned int spin             = indices[4];
  unsigned int cyclic_index     = indices[5]; 

  fptype rMassSq = (PAIR_12 == cyclic_index ? m12 : (PAIR_13 == cyclic_index ? m13 : m23));
  fptype frFactor = 1;

  resmass *= resmass; 
  // Calculate momentum of the two daughters in the resonance rest frame; note symmetry under interchange (dm1 <-> dm2). 
  fptype measureDaughterMoms = twoBodyCMmom(rMassSq, (PAIR_23 == cyclic_index ? daug2Mass : daug1Mass), (PAIR_12 == cyclic_index ? daug2Mass : daug3Mass));
  fptype nominalDaughterMoms = twoBodyCMmom(resmass, (PAIR_23 == cyclic_index ? daug2Mass : daug1Mass), (PAIR_12 == cyclic_index ? daug2Mass : daug3Mass));

  if (0 != spin) {
    frFactor =  dampingFactorSquare(nominalDaughterMoms, spin, meson_radius);
    frFactor /= dampingFactorSquare(measureDaughterMoms, spin, meson_radius); 
  }
  
  // Implement Gou-Sak:

  //fptype D = resmass*(1.0 + dFun(resmass, daug2Mass, daug3Mass) * reswidth/SQRT(resmass));
  fptype D = (1.0 + dFun(resmass, daug2Mass, daug3Mass) * reswidth/SQRT(resmass));
  fptype E = resmass - rMassSq + fsFun(rMassSq, resmass, reswidth, daug2Mass, daug3Mass);
  fptype F = SQRT(resmass) * reswidth * POW(measureDaughterMoms / nominalDaughterMoms, 2.0*spin + 1) * frFactor;

  D       /= (E*E + F*F);
  devcomplex<fptype> retur(D*E, D*F); // Dropping F_D=1
  retur *= SQRT(frFactor);
  retur *= spinFactor(spin, motherMass, daug1Mass, daug2Mass, daug3Mass, m12, m13, m23, cyclic_index);

  return retur; 
}


  EXEC_TARGET devcomplex<fptype> lass (fptype m12, fptype m13, fptype m23, unsigned int* indices) {
  fptype motherMass             = functorConstants[indices[1]+0];
  fptype daug1Mass              = functorConstants[indices[1]+1];
  fptype daug2Mass              = functorConstants[indices[1]+2];
  fptype daug3Mass              = functorConstants[indices[1]+3];
  fptype meson_radius           = functorConstants[indices[1]+4];

  fptype resmass                = hipArray[indices[2]];
  fptype reswidth               = hipArray[indices[3]];
//  fptype lass_a                 = hipArray[indices[4]];
//  fptype lass_r                 = hipArray[indices[5]];
//  fptype lass_B                 = hipArray[indices[6]];
//  fptype lass_phiB              = hipArray[indices[7]];
//  fptype lass_R                 = hipArray[indices[8]];
//  fptype lass_phiR              = hipArray[indices[9]];
  unsigned int spin             = indices[10];
  unsigned int cyclic_index     = indices[11];

  fptype rMassSq = (PAIR_12 == cyclic_index ? m12 : (PAIR_13 == cyclic_index ? m13 : m23));
  fptype frFactor = 1;

  resmass *= resmass;
  // Calculate momentum of the two daughters in the resonance rest frame; note symmetry under interchange (dm1 <-> dm2).
  
  fptype measureDaughterMoms = twoBodyCMmom(rMassSq, (PAIR_23 == cyclic_index ? daug2Mass : daug1Mass), (PAIR_23 == cyclic_index ? daug3Mass : daug2Mass));
  fptype nominalDaughterMoms = twoBodyCMmom(resmass, (PAIR_23 == cyclic_index ? daug2Mass : daug1Mass), (PAIR_23 == cyclic_index ? daug3Mass : daug2Mass));

  if (0 != spin) {
    frFactor =  dampingFactorSquare(nominalDaughterMoms, spin, meson_radius);
    frFactor /= dampingFactorSquare(measureDaughterMoms, spin, meson_radius);
  }

  //Implement LASS:

  fptype q = measureDaughterMoms;
  fptype g = reswidth * POW(measureDaughterMoms / nominalDaughterMoms, 2.0*spin + 1) * frFactor / SQRT(rMassSq);
  //fptype g = reswidth * POW(measureDaughterMoms / nominalDaughterMoms, 2.0*spin + 1) * frFactor * ( SQRT(resmass) / SQRT(rMassSq)); // as in PDG
  fptype lass_a    = 0.22357;
  fptype lass_r    = -15.042;
  fptype lass_R    = 1; // ?
  fptype lass_phiR = 1.10644;
  fptype lass_B    = 0.614463;
  fptype lass_phiB = -0.0981907;

  // background phase motion
  fptype cot_deltaB = (1.0 / (lass_a*q)) + 0.5*lass_r*q;
  fptype qcot_deltaB = (1.0 / lass_a) + 0.5*lass_r*q*q;

  // calculate resonant part
  devcomplex<fptype> expi2deltaB = devcomplex<fptype>(qcot_deltaB,q)/devcomplex<fptype>(qcot_deltaB,-q);
  devcomplex<fptype>  resT = devcomplex<fptype>(cos(lass_phiR+2*lass_phiB),sin(lass_phiR+2*lass_phiB))*lass_R;

  devcomplex<fptype> prop = devcomplex<fptype>(1, 0)/devcomplex<fptype>(resmass-rMassSq, SQRT(resmass)*g);
  //devcomplex<fptype> prop = devcomplex<fptype>(1, 0)/devcomplex<fptype>(resmass-rMassSq, -SQRT(resmass)*g); // as in EvtGen and Papers
  //resT *= prop*(SQRT(resmass)*reswidth/nominalDaughterMoms)*expi2deltaB;
  resT *= prop*(resmass*reswidth/nominalDaughterMoms)*expi2deltaB;

  // calculate bkg part
  resT += devcomplex<fptype>(cos(lass_phiB),sin(lass_phiB))*lass_B*q*(cos(lass_phiB)+cot_deltaB*sin(lass_phiB))/devcomplex<fptype>(qcot_deltaB,-q);
  //resT += devcomplex<fptype>(cos(_phiB),sin(_phiB))*_B*(cos(_phiB)+cot_deltaB*sin(_phiB))*SQRT(rMassSq)/devcomplex<fptype>(qcot_deltaB,-q);

  resT *= SQRT(frFactor);
  resT *= spinFactor(spin, motherMass, daug1Mass, daug2Mass, daug3Mass, m12, m13, m23, cyclic_index);

  return resT;
}

EXEC_TARGET devcomplex<fptype> nonres (fptype m12, fptype m13, fptype m23, unsigned int* indices) {
  return devcomplex<fptype>(1, 0); 
}


EXEC_TARGET void getAmplitudeCoefficients (devcomplex<fptype> a1, devcomplex<fptype> a2, fptype& a1sq, fptype& a2sq, fptype& a1a2real, fptype& a1a2imag) {
  // Returns A_1^2, A_2^2, real and imaginary parts of A_1A_2^*
  a1sq = a1.abs2();
  a2sq = a2.abs2();
  a1 *= conj(a2);
  a1a2real = a1.real;
  a1a2imag = a1.imag; 
}

 MEM_DEVICE resonance_function_ptr ptr_to_RBW = plainBW;
 MEM_DEVICE resonance_function_ptr ptr_to_GOUSAK = gouSak; 
 MEM_DEVICE resonance_function_ptr ptr_to_GAUSSIAN = gaussian;
 MEM_DEVICE resonance_function_ptr ptr_to_NONRES = nonres;
 MEM_DEVICE resonance_function_ptr ptr_to_LASS = lass;
 MEM_DEVICE resonance_function_ptr ptr_to_kMatrix = Get_kMatrix; 

 ResonancePdf::ResonancePdf (string name,
						Variable* ar,             
                                                Variable* ai,             
						Variable* Spr0,
                                                unsigned int term,
						unsigned int sp,         	
						unsigned int cyc)               
  :GooPdf(0,name)
  ,amp_real(ar)                                                         
  ,amp_imag(ai)
{
  vector <unsigned int> pindices; 
  pindices.push_back(0); 
  pindices.push_back(registerParameter(Spr0));
  pindices.push_back(term);
  pindices.push_back(sp);
  pindices.push_back(cyc); 


 
  GET_FUNCTION_ADDR(ptr_to_kMatrix);
  initialise(pindices); 
}

ResonancePdf::ResonancePdf (string name, 
						Variable* ar, 
						Variable* ai, 
						Variable* mass, 
						Variable* width, 
						unsigned int sp, 
						unsigned int cyc) 
  : GooPdf(0, name)
  , amp_real(ar)
  , amp_imag(ai)
{
  vector<unsigned int> pindices; 
  pindices.push_back(0); 
  // Making room for index of decay-related constants. Assumption:
  // These are mother mass and three daughter masses in that order.
  // They will be registered by the object that uses this resonance,
  // which will tell this object where to find them by calling setConstantIndex. 

  pindices.push_back(registerParameter(mass));
  pindices.push_back(registerParameter(width)); 
  pindices.push_back(sp);
  pindices.push_back(cyc); 

  GET_FUNCTION_ADDR(ptr_to_RBW);
  initialise(pindices); 
}

ResonancePdf::ResonancePdf (string name, 
						Variable* ar, 
						Variable* ai, 
						unsigned int sp, 
						Variable* mass, 
						Variable* width, 
						unsigned int cyc) 
  : GooPdf(0, name)
  , amp_real(ar)
  , amp_imag(ai)
{
  // Same as BW except for function pointed to. 
  vector<unsigned int> pindices; 
  pindices.push_back(0); 
  pindices.push_back(registerParameter(mass));
  pindices.push_back(registerParameter(width)); 
  pindices.push_back(sp);
  pindices.push_back(cyc); 

  GET_FUNCTION_ADDR(ptr_to_GOUSAK);
  initialise(pindices); 
} 


ResonancePdf::ResonancePdf (string name,
                                                Variable* ar,
                                                Variable* ai,
						Variable* mass,
//						Variable* a,
//						Variable* r,
//						Variable* B,
//						Variable* phiB,
//						Variable* R,
//						Variable* phiR,
                                                unsigned int sp,
                                                Variable* width,
                                                unsigned int cyc)
  : GooPdf(0, name)
  , amp_real(ar)
  , amp_imag(ai)
{
  // Same as BW except for function pointed to.
  vector<unsigned int> pindices;
  pindices.push_back(0);
  pindices.push_back(registerParameter(mass));
  //pindices.push_back(registerParameter(a));
  //pindices.push_back(registerParameter(r));
  //pindices.push_back(registerParameter(B));
  //pindices.push_back(registerParameter(phiB));
  //pindices.push_back(registerParameter(R));
  //pindices.push_back(registerParameter(phiR));
  pindices.push_back(sp);
  pindices.push_back(registerParameter(width));
  pindices.push_back(cyc);

  GET_FUNCTION_ADDR(ptr_to_LASS);
  initialise(pindices);
}

ResonancePdf::ResonancePdf (string name, 
						Variable* ar, 
						Variable* ai) 
  : GooPdf(0, name)
  , amp_real(ar)
  , amp_imag(ai)
{
  vector<unsigned int> pindices; 
  pindices.push_back(0); 
  // Dummy index for constants - won't use it, but calling 
  // functions can't know that and will call setConstantIndex anyway. 
  GET_FUNCTION_ADDR(ptr_to_NONRES);
  initialise(pindices); 
}

ResonancePdf::ResonancePdf (string name,
						Variable* ar, 
						Variable* ai,
						Variable* mean, 
						Variable* sigma,
						unsigned int cyc) 
  : GooPdf(0, name)
  , amp_real(ar)
  , amp_imag(ai)
{
  vector<unsigned int> pindices; 
  pindices.push_back(0); 
  // Dummy index for constants - won't use it, but calling 
  // functions can't know that and will call setConstantIndex anyway. 
  pindices.push_back(registerParameter(mean));
  pindices.push_back(registerParameter(sigma)); 
  pindices.push_back(cyc); 

  GET_FUNCTION_ADDR(ptr_to_GAUSSIAN);
  initialise(pindices); 

}


